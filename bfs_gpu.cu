#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

#define INF 1e9

__global__ void bfs_gpu(int* edges, int* offsets, int* distances, int* changed, int num_nodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_nodes || distances[tid] == INF) return;

    int start = offsets[tid];
    int end = offsets[tid + 1];

    for (int i = start; i < end; i++) {
        int neighbor = edges[i];
        if (atomicMin(&distances[neighbor], distances[tid] + 1) > distances[tid] + 1) {
            *changed = 1;
        }
    }
}

void bfs_cuda(const std::vector<std::vector<int>>& graph, int start_node) {
    int num_nodes = graph.size();

    std::vector<int> edges;
    std::vector<int> offsets(num_nodes + 1, 0);

    for (int i = 0; i < num_nodes; ++i) {
        offsets[i + 1] = offsets[i] + graph[i].size();
        edges.insert(edges.end(), graph[i].begin(), graph[i].end());
    }

    int* d_edges;
    int* d_offsets;
    int* d_distances;
    int* d_changed;

    std::vector<int> distances(num_nodes, INF);
    distances[start_node] = 0;

    hipMalloc(&d_edges, edges.size() * sizeof(int));
    hipMalloc(&d_offsets, offsets.size() * sizeof(int));
    hipMalloc(&d_distances, num_nodes * sizeof(int));
    hipMalloc(&d_changed, sizeof(int));

    hipMemcpy(d_edges, edges.data(), edges.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), offsets.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_distances, distances.data(), num_nodes * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256); //each block contains 256 threads
    dim3 blocksPerGrid((num_nodes + threadsPerBlock.x - 1) / threadsPerBlock.x);

    auto start_time = std::chrono::high_resolution_clock::now();

    while (true) {
        int h_changed = 0;
        hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice);

        bfs_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_edges, d_offsets, d_distances, d_changed, num_nodes);
        hipDeviceSynchronize();

        hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost);
        if (!h_changed) break;
    }

    auto end_time = std::chrono::high_resolution_clock::now();

    hipMemcpy(distances.data(), d_distances, num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_nodes; i++) {
        if (distances[i] != INF)
            std::cout << "Node " << i << ": Distance from start: " << distances[i] << std::endl;
    }

    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "GPU BFS Time: " << elapsed.count() << " seconds\n";

    hipFree(d_edges);
    hipFree(d_offsets);
    hipFree(d_distances);
    hipFree(d_changed);
}

int main() {
    int num_nodes = 1000000;
    int num_edges = 10000000;
    std::vector<std::vector<int>> graph(num_nodes);

    srand(42);
    for (int i = 0; i < num_edges; i++) {
        int u = rand() % num_nodes;
        int v = rand() % num_nodes;
        if (u != v) {
            graph[u].push_back(v);
            graph[v].push_back(u);
        }
    }

    bfs_cuda(graph, 0);
    return 0;
}
